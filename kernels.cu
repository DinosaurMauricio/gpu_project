#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

#ifndef TILE_DIM
#define TILE_DIM 32
#endif

__global__ void transposeWithTiledPartition(DATA_TYPE *odata, const DATA_TYPE *idata, int matrixSize)
{
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<TILE_DIM> tile32 = cg::tiled_partition<TILE_DIM>(block);
    
    __shared__ DATA_TYPE tile[TILE_DIM][TILE_DIM + 1];

    int x = blockIdx.x * TILE_DIM + tile32.thread_rank();
    int y = blockIdx.y * TILE_DIM + tile32.meta_group_rank();

    // Load data into shared memory
    tile[tile32.meta_group_rank()][tile32.thread_rank()] = idata[y * matrixSize + x];


    block.sync();

    x = blockIdx.y * TILE_DIM + tile32.thread_rank();
    y = blockIdx.x * TILE_DIM + tile32.meta_group_rank();

    // Store transposed data from shared memory to global memory
    odata[y * matrixSize + x] = tile[tile32.thread_rank()][tile32.meta_group_rank()];
}

__global__ void transposeTileKernelChild(DATA_TYPE *odata, const DATA_TYPE *idata, int xOffset, int yOffset, int matrixSize) {
    __shared__ DATA_TYPE tile[TILE_DIM][TILE_DIM + 1];

    int x = xOffset + threadIdx.x;
    int y = yOffset + threadIdx.y;

    if (x < matrixSize && y < matrixSize) {
        tile[threadIdx.y][threadIdx.x] = idata[y * matrixSize + x];
    }

    __syncthreads();

    x = yOffset + threadIdx.x;
    y = xOffset + threadIdx.y;

    if (x < matrixSize && y < matrixSize) {
        odata[y * matrixSize + x] = tile[threadIdx.x][threadIdx.y];
    }
}

__global__ void transposeKernelParent(DATA_TYPE *odata, const DATA_TYPE *idata, int matrixSize) {
    int xTile = blockIdx.x * TILE_DIM;
    int yTile = blockIdx.y * TILE_DIM;

    if (xTile < matrixSize && yTile < matrixSize) {
        transposeTileKernelChild<<<1, dim3(TILE_DIM, TILE_DIM)>>>(odata, idata, xTile, yTile, matrixSize);
    }
}

