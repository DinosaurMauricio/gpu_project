#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;
extern "C" {
#include "my_library.h"
}


#define BLOCK_SIZE 4
#define MATRIX_SIZE 4


#ifndef TILE_DIM
#define TILE_DIM 4
#endif

#ifndef BLOCK_ROWS
#define BLOCK_ROWS 4
#endif


__global__ void transpose_block_2d_matrix(float* mat, size_t sx, size_t sy)
{
    constexpr size_t block_size = TILE_DIM;
    cg::thread_block block = cg::this_thread_block();
    auto tile = cg::tiled_partition<block_size>(block);
    size_t ix = tile.thread_rank();

    // Calculate global indices
    size_t block_x = blockIdx.x * block_size;
    size_t block_y = blockIdx.y * block_size;

    // Define shared memory to hold the tile
    __shared__ float tile_shared[block_size][block_size];

    // Load data into shared memory
    if (block_x + ix < sx && block_y + tile.thread_rank() < sy)
    {
        for (size_t iy = 0; iy < block_size; ++iy)
        {
            if (block_y + iy < sy)
            {
                tile_shared[ix][iy] = mat[(block_x + ix) + (block_y + iy) * sx];
            }
        }
    }

    block.sync();

    // Transpose data in shared memory and write back to global memory
    if (block_x + tile.thread_rank() < sx && block_y + ix < sy)
    {
        for (size_t iy = 0; iy < block_size; ++iy)
        {
            if (block_y + tile.thread_rank() < sy)
            {
                mat[(block_y + ix) + (block_x + iy) * sx] = tile_shared[iy][ix];
            }
        }
    }
}


__global__ void transpose_block(float* mat, size_t sx, size_t sy)
{
    constexpr size_t size = BLOCK_SIZE;
    cg::thread_block block = cg::this_thread_block();
    auto tile = cg::tiled_partition<size>(block );
    auto ix = tile.thread_rank();

    float col[size];
    for (size_t iy = 0; iy < size; ++iy)
    {
        col[iy] = mat[ix + iy * sx];
    }

    block.sync();

    auto val = [&tile, &col](int ix, int iy) { 
        return tile.shfl(col[iy], ix); 
    };

    for (size_t idx = 1; idx < size; ++idx){
        size_t iy = threadIdx.x^idx;
        float result = val(iy, iy);
        
        mat[ix + iy * sx] = result;
        
        }
}

void print_mat(float* mat, size_t sx, size_t sy)
{
    printf("{\n");
    for (size_t iy = 0; iy < sy; ++iy)
    {
        printf("\t{ ");
        for (size_t ix = 0; ix < sx; ++ix)
            printf("%6.1f, ", mat[ix + iy * sx]);
        printf("},\n");
    }
    printf("}\n");
}

int main()
{
    hipEvent_t startEvent, stopEvent;

    // events for timing
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);


    constexpr size_t sx = MATRIX_SIZE; // Size of the matrix
    constexpr size_t sy = MATRIX_SIZE;
    float* mat;
    int numberOfTests = 100;

    hipMallocManaged(&mat, sx * sy * sizeof(float));

    for (size_t iy = 0; iy < sy; ++iy)
        for (size_t ix = 0; ix < sx; ++ix)
            mat[ix + sx * iy] = ix + sx * iy;

    print_mat(mat, sx, sy);

    dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid_size((sx + block_size.x - 1) / block_size.x, (sy + block_size.y - 1) / block_size.y);

    //dim3 grid_size(sx / TILE_DIM, sy / TILE_DIM, 1);
    //dim3 block_size(TILE_DIM, BLOCK_SIZE, 1);

    printf("Block size: %d %d, Tile size: %d %d\n", TILE_DIM, BLOCK_ROWS, TILE_DIM, TILE_DIM);
    printf("dimGrid: %d %d %d. dimBlock: %d %d %d\n",
           grid_size.x, grid_size.y, grid_size.z, block_size.x, block_size.y, block_size.z);

    double total_bw = 0;
    float total_ms = 0;
    const int repeat = 5;


    for (int i = 0; i < repeat; i++) {
            unsigned long long effective_bw;
            float ms;

            // Warm up
            transpose_block<<<grid_size, block_size>>>(mat, sx, sy);
            hipEventRecord(startEvent, 0);
            for (int i = 0; i < numberOfTests; i++)
                transpose_block<<<grid_size, block_size>>>(mat, sx, sy);
            hipEventRecord(stopEvent, 0);
            hipEventSynchronize(stopEvent);
            hipEventElapsedTime(&ms, startEvent, stopEvent);
            //checkCuda(hipMemcpy(h_cdata, d_cdata, memory_size, hipMemcpyDeviceToHost));
            effective_bw = calculate_effective_bandwidth(sx * sy, numberOfTests, ms);

            printf("%20llu %20.2f ms\n", effective_bw, ms);
            total_bw += effective_bw;
            total_ms += ms;
    }

    double avg_bw = total_bw / repeat;
    double avg_ms = total_ms / repeat;

    printf("Average Bandwidth (GB/s): %llu\n", avg_bw);
    printf("Average Time (ms): %.2f\n", avg_ms);

    print_mat(mat, sx, sy);


    hipFree(mat);
}
