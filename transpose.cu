#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
namespace cg = cooperative_groups;



#ifndef TILE_DIM
#define TILE_DIM 32
#endif

#ifndef MATRIX_SIZE
#define MATRIX_SIZE 1024
#endif

#define NUMBER_OF_TESTS 100

#ifdef DATA_TYPE_FLOAT
    #define CUBLAS_Geam hipblasSgeam
#elif defined(DATA_TYPE_DOUBLE)
    #define CUBLAS_Geam hipblasDgeam
#else
    #error "Define DATA_TYPE_FLOAT or DATA_TYPE_DOUBLE"
#endif
#include "kernels.cu"
#include "cuda_utils.cu"


int main()
{
    const unsigned long long memory_size = MATRIX_SIZE * MATRIX_SIZE * sizeof(DATA_TYPE);

    int devID = 0;
    hipDeviceProp_t deviceProp;

    getDeviceProperties(devID, deviceProp);

    if (!checkMemorySize(memory_size, deviceProp)) {
        exit(EXIT_FAILURE);
    }

    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    DATA_TYPE *h_idata, *h_odata;
    h_idata = (DATA_TYPE*)malloc(memory_size);
    h_odata = (DATA_TYPE*)malloc(memory_size);

    initializeMatrixValues(h_idata, MATRIX_SIZE);

    /*printf("Original \n ");
    printMatrix(h_idata, MATRIX_SIZE);
    printf("\n");
*/
    DATA_TYPE *d_idata, *d_odata;
    hipMalloc(&d_idata, memory_size);
    hipMalloc(&d_odata, memory_size);
    hipMemcpy(d_idata, h_idata, memory_size, hipMemcpyHostToDevice);

    dim3 grid(MATRIX_SIZE / TILE_DIM, MATRIX_SIZE / TILE_DIM, 1);
    dim3 threads(TILE_DIM, TILE_DIM, 1);

    printf("dimGrid: %d %d %d. dimThreads: %d %d %d\n",
           grid.x, grid.y, grid.z, threads.x, threads.y, threads.z);

    printf("*****************************************************************************\n");

    printf("%25s", "transposeWithTiledPartition\n");
    runTransposeKernel(transposeWithTiledPartition, grid, threads, d_odata, d_idata, memory_size, NUMBER_OF_TESTS, startEvent, stopEvent);

    printf("*****************************************************************************\n");

    printf("%25s", "transposeKernelParent\n");
    runTransposeKernel(transposeKernelParent,grid, threads, d_odata, d_idata, memory_size, NUMBER_OF_TESTS, startEvent, stopEvent);

     // CUBLAS operations
    printf("*****************************************************************************\n");
    printf("%25s", "cuBLAS\n");
    DATA_TYPE *d_A, *d_B;
    DATA_TYPE *h_A = (DATA_TYPE *)malloc(MATRIX_SIZE * MATRIX_SIZE * sizeof(DATA_TYPE));

    for (int i = 0; i < MATRIX_SIZE * MATRIX_SIZE; i++) {
        h_A[i] = static_cast<DATA_TYPE>(rand()) / RAND_MAX;
    }

    hipMalloc((void **)&d_A, MATRIX_SIZE * MATRIX_SIZE * sizeof(DATA_TYPE));
    hipMalloc((void **)&d_B, MATRIX_SIZE * MATRIX_SIZE * sizeof(DATA_TYPE));
    hipMemcpy(d_A, h_A, MATRIX_SIZE * MATRIX_SIZE * sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    runCUBLASOperations(d_A, d_B, NUMBER_OF_TESTS, startEvent, stopEvent);
    
    hipFree(d_idata);
    hipFree(d_odata);
    free(h_idata);
    free(h_odata);

    return 0;
}
